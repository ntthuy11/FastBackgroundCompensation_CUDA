#include "hip/hip_runtime.h"
#include <cutil_inline.h>

/*#define N_BINS			16
#define DIV_VAL_HIST		16	// GRAY_LEVELS / 16 = 256 / 16

#define N_THREADS_SCALE		16
#define VER_HIST_SMEM_SIZE	256 // N_BINS x N_THREADS_SCALE = 16 x 16
*/

#define N_BINS				8
#define DIV_VAL_HIST		32	

#define N_THREADS_SCALE		16
#define VER_HIST_SMEM_SIZE	128


unsigned int *d_im0, *d_im1;
float *h_scales, *d_scales;
float *h_diffScaleVerHists, *d_diffScaleVerHists;

float sy;
float syDiff;


// ========================================= PRIVATE UTILS =========================================


void checkErrorScale(const char *msg) {
    hipError_t err = hipGetLastError();
    if(hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);    
	}
}


// ========================================= KERNELS =========================================


__global__ void kernel_calcDiffScaleVerHists(float *diffScaleVerHists, unsigned int *im0, unsigned int *im1, int imH, int imW, 
											 float *scales, int nScales) { // scale < 1: zoom out, scale > 1: zoom in

	int globalIdx	= __umul24(blockIdx.x, blockDim.x) + threadIdx.x;

	int iThScale	= globalIdx / imW;
	int iThColumn	= globalIdx % imW;
	float scale		= scales[iThScale];


	// ---------- define shared memories ----------
	__shared__ int smem_verHist0[VER_HIST_SMEM_SIZE]; // with size = N_BINS * VER_HIST_N_THREADS_SCALE = 16 bins * 16 threads = 256 (one thread processes one column)
	__shared__ int smem_verHist1[VER_HIST_SMEM_SIZE];

	__shared__ float smem_diffVerHist[N_THREADS_SCALE];


	// init shared memories to 0
	#pragma unroll
	for(int i = 0; i < N_BINS; i++) {
		int smemPos = __mul24(i, N_THREADS_SCALE) + threadIdx.x;
		smem_verHist0[smemPos] = 0;
		smem_verHist1[smemPos] = 0;
	}
	__syncthreads();



	// ---------- calculate vertical histograms of 2 images and store to shared memories ----------
	int interpolatedImH = -1;
	int startPos = -1;

	// determine zoom out or zoom in
	float scaleUp = 1 - (scale - 1);

	if(scale < 1) { // zoom out
		interpolatedImH = int(imH * scale);		// for im0			// interpolatedImgH < imH
		startPos = int(imH * (1 - scale)/2);	// for im1
	} else { // zoom in
		startPos = int(imH * (1 - scaleUp)/2);	// for im0
		interpolatedImH = int(imH * scaleUp); // for img1	
	}

	// calc hist.
	int tmp = __mul24(startPos, imW) + iThColumn;

	#pragma unroll
	for(int i = 0; i < interpolatedImH; i++) {
		int interpolatedI = -1;

		if(scale < 1) { // zoom out
			interpolatedI = int(i/scale); // i * (1 / scale)		// interpolatedI > i
		} else { // zoom in
			interpolatedI = int(i/scaleUp);
		}

		int imPos0 = __mul24(interpolatedI, imW) + iThColumn;		
		int imPos1 = __mul24(i, imW) + tmp;
		//int imgPos1 = (i + startPos) * imgW + idx; // = i*imgW + startPos*imgW + idx = i*imgW + tmp;

		int binIdx0 = im0[imPos0] / DIV_VAL_HIST;		int smemPos0 = __mul24(binIdx0, N_THREADS_SCALE) + threadIdx.x;		smem_verHist0[smemPos0]++;
		int binIdx1 = im1[imPos1] / DIV_VAL_HIST;		int smemPos1 = __mul24(binIdx1, N_THREADS_SCALE) + threadIdx.x;		smem_verHist1[smemPos1]++;
	}
	__syncthreads();



	// ---------- calculate the difference btw vertical histograms and store to global memory ----------
	int diff = 0;

	#pragma unroll
	for(int i = 0; i < N_BINS; i++) {
		int smemPos = __mul24(i, N_THREADS_SCALE) + threadIdx.x;
		diff += abs(smem_verHist0[smemPos] - smem_verHist1[smemPos]);
	}
	smem_diffVerHist[threadIdx.x] = 1 - diff*0.5/imH;
	__syncthreads();

	diffScaleVerHists[globalIdx] = smem_diffVerHist[threadIdx.x];
}


// ==========================================================================================================================


void initScale(int *img0, int *img1, int imH, int imW,
			   double scaleFrom, double scaleTo, double scaleRes, int nScales) {
    
	int fourBytes = sizeof(int);
	

	// ----- init arrays representing for 2 input images -----
	size_t memSizeOfImage = imH * imW * fourBytes;
	hipMalloc((void **) &d_im0, memSizeOfImage);					hipMemcpy(d_im0, img0, memSizeOfImage, hipMemcpyHostToDevice);
	hipMalloc((void **) &d_im1, memSizeOfImage);					hipMemcpy(d_im1, img1, memSizeOfImage, hipMemcpyHostToDevice);	


	// ----- init an array of scales -----
	size_t memSizeOfNScales = nScales * fourBytes;
	h_scales = (float *)malloc(memSizeOfNScales);

	int count = 0;
	for(double i = scaleFrom; i <= scaleTo; i += scaleRes) // i < 1: zoom out, i > 1: zoom in
		h_scales[count++] = float(i);

	hipMalloc((void **) &d_scales, memSizeOfNScales);
	hipMemcpy(d_scales, h_scales, memSizeOfNScales, hipMemcpyHostToDevice);


	// ----- init array for the different btw 2 vertical histograms -----
	size_t memSizeOfDiffScaleVerHists = nScales * imW * fourBytes;
	hipMalloc((void **) &d_diffScaleVerHists, memSizeOfDiffScaleVerHists);
	h_diffScaleVerHists = (float *)malloc(memSizeOfDiffScaleVerHists);
}


void calcDiffScaleVerHists(int imH, int imW, int nScales) {
	int nThreads = N_THREADS_SCALE;
	int nBlocks = (imW / nThreads) * nScales; // imageWidth chia het cho 16

	kernel_calcDiffScaleVerHists<<<nBlocks, nThreads>>>(d_diffScaleVerHists, d_im0, d_im1, imH, imW, d_scales, nScales);	checkErrorScale("kernel_calcDiffScaleVerHists");
	hipDeviceSynchronize();																								checkErrorScale("kernel_calcDiffScaleVerHists hipDeviceSynchronize");

	size_t memSizeOfDiffScaleVerHists = nScales * imW * sizeof(float);
	hipMemcpy(h_diffScaleVerHists, d_diffScaleVerHists, memSizeOfDiffScaleVerHists, hipMemcpyDeviceToHost);				checkErrorScale("copy from DEV to HOST");
}


float calcGlobalDiffScaleVerHist(int imW, int iThScale) {
	float diff = 0;
	for(int i = 0; i < imW; i++) 
		diff += h_diffScaleVerHists[iThScale * imW + i];
	return diff / imW;
}


// --------------------------------------------------------------------


/*void estimateScale(int *img0, int *img1, int imH, int imW,
				   double scaleFrom, double scaleTo, double scaleRes) {

	// initialization
	int nScales = int((scaleTo - scaleFrom) / scaleRes + 1);
	initScale(img0, img1, imH, imW, scaleFrom, scaleTo, scaleRes, nScales); // co phan nay la ton > 30 % thoi gian

	// calculate sy
	calcDiffScaleVerHists(imH, imW, nScales);

	syDiff = -1.0;
	for(int iThScale = 0; iThScale < nScales; iThScale++) { // i < 1: zoom out, i > 1: zoom in
		float diff = calcGlobalDiffScaleVerHist(imW, iThScale);
		if (syDiff < diff) {
			syDiff = diff;
			sy = float(scaleFrom + iThScale*scaleRes);
		}
	}
}*/


void releaseScale() {
	hipFree(d_im0);			hipFree(d_im1);
	free(h_scales);				hipFree(d_scales);
	free(h_diffScaleVerHists);	hipFree(d_diffScaleVerHists);
}


// --------------------------------------------------------------------


extern "C" float getSy() {	return sy;	}


extern "C" float runScaleEst(int *img0, int *img1, int imH, int imW,
							 double scaleFrom, double scaleTo, double scaleRes) {

	// ---------------------- initialization ----------------------
	int nScales = int((scaleTo - scaleFrom) / scaleRes + 1);
	initScale(img0, img1, imH, imW, scaleFrom, scaleTo, scaleRes, nScales); // co phan nay la ton > 30 % thoi gian
	// ------------------------------------------------------------


		// computation time measure
		hipEvent_t timeStart, timeStop;

		float elapsedTime;
		hipEventCreate(&timeStart);
		hipEventCreate(&timeStop);
		hipEventRecord(timeStart, 0);
	

	// ---------------------- calculate sy ----------------------
	calcDiffScaleVerHists(imH, imW, nScales);

	syDiff = -1.0;
	for(int iThScale = 0; iThScale < nScales; iThScale++) { // i < 1: zoom out, i > 1: zoom in
		float diff = calcGlobalDiffScaleVerHist(imW, iThScale);
		if (syDiff < diff) {
			syDiff = diff;
			sy = float(scaleFrom + iThScale*scaleRes);
		}
	}
	// -----------------------------------------------------------

	
		hipEventRecord(timeStop, 0);
		hipEventSynchronize(timeStop);
		hipEventElapsedTime(&elapsedTime, timeStart, timeStop);
		hipEventDestroy(timeStart);
		hipEventDestroy(timeStop);

		// print results to file
		FILE *fp;
		fp = fopen("result.txt", "a");
		//fprintf(fp, "sy=%f, syDiff=%f,   t=%f \n", getSy(), syDiff, elapsedTime);
		fprintf(fp, "sy=%1.3f, t=%1.3f   ", getSy(), elapsedTime);
		fclose(fp);


	releaseScale();

	return elapsedTime;
}